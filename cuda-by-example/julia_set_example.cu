#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define DIM 1000

// https://devtalk.nvidia.com/default/topic/836926/how-to-compile-codes-on-cuda-opengl-interop-from-the-book-cuda-by-example-by-jason-sanders-amp-edw/

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y ) {
    const float scale = 1.2;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel( unsigned char *ptr ) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * (1-juliaValue);
    ptr[offset*4 + 1] = 255 * (1-juliaValue);
    ptr[offset*4 + 2] = 255 * (1-juliaValue);
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;

    dim3 grid(DIM,DIM);
    kernel<<<grid,1>>>(dev_bitmap);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
                              
    hipFree(dev_bitmap);
                              
    bitmap.display_and_exit();
}