#include "hip/hip_runtime.h"
#include <stdio.h>
#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n) // __global__ �� �� �Լ��� GPU������ ���� �� ������ �˷���
{
	int i = threadIdx.x; // read only variable

	if (i < n)
		c[i] = a[i] * b[i];
	//int i; // ���� for������ �ۼ��ϸ� �̷��� �ȴ�.
	//for (i = 0; i < n; ++i)
	//	c[i] = a[i] + b[i];
}


int main()
{
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int)); // cuda ���� �޸� �Ҵ��ϴ� ���
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++) // ��Ʈ������ CPU�� �̿��ؼ� �ʱ�ȭ �Ѵ�. (������ �����͸� �������� ������ ����) 
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	VectorAdd <<<1, SIZE>>> (a, b, c, SIZE);

	hipDeviceSynchronize();

	int count;
	hipGetDeviceCount(&count);

	printf("The number of GPU devices is %d\n", count);
	hipDeviceProp_t prop;
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);
		printf(" --- General Information for device %d ---\n", i);
		printf(" Name : %s\n", prop.name);
		printf(" Compute capability: %d.%d\n", prop.major, prop.minor);
		printf(" Clock rate : %d \n", prop.clockRate);
		
		printf(" Total global memory : %ld MB\n", prop.totalGlobalMem/(1024*1024));
		printf(" Multiprocessor count : %d\n", prop.multiProcessorCount);
	}



	printf("\n\n");
	printf("CUDA Matrix addition example\n");
	for (int i = 0; i < 10; ++i)
		printf(" c[%d] = %d\n", i, c[i]);

	hipFree(a); // free ��ſ� hipFree�� GPU�� �Ҵ��� �޸� ����
	hipFree(b);
	hipFree(c);

	return 0;
}